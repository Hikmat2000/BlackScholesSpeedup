#include "hip/hip_runtime.h"
#include "black_scholes.h"
#include "util.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

__managed__ double stddev;

__global__ void black_scholes_stddev (void* the_args)
{

  black_scholes_args_t* args = (black_scholes_args_t*) the_args;
  const double mean = args->mean;
  const int M = args->M;
  double variance = 0.0;
  int k = blockIdx.x * blockDim.x + threadIdx.x;

  if(k<M)
  {
   const double diff = args->trials[k] - mean;
   variance += diff * diff / (double) M;
  }

  args->variance = variance;
  stddev=sqrt(variance);

}


__global__ void black_scholes_iterate (void* the_args)
{

  black_scholes_args_t* args = (black_scholes_args_t*) the_args;

  const int S = args->S;
  const int E = args->E;
  const int M = args->M;
  const double r = args->r;
  const double sigma = args->sigma;
  const double T = args->T;
 // hiprandStateMtgp32* state=args->state;
  double* trials = args->trials;
  double mean = 0.0;
  //double* devResults=args

int i = blockIdx.x * blockDim.x + threadIdx.x;
int k = blockIdx.x * blockDim.x + threadIdx.x;
if(i<M)
{
 //  devResults[i] = hiprand(&state[blockIdx.x]);
  const double current_value = S * exp ( (r - (sigma*sigma) / 2.0) * T + sigma * sqrt (T) *1);
  trials[k] = exp (-r * T) * ((current_value - E < 0.0) ? 0.0 : current_value - E);
   mean += trials[k]/ (double) M; 
   __syncthreads();
  args->mean = mean;
}  
}



void myfunction(confidence_interval_t* interval,
   const double S, const double E, const double r, const double sigma, const double T, const int M,const int n)
{

  black_scholes_args_t args;
  double mean = 0.0;
  double conf_width = 0.0;
  double* trials = NULL;

  assert (M > 0);
  trials = (double*) malloc (M * sizeof (double));
  assert (trials != NULL);

  args.S = S;
  args.E = E;
  args.r = r;
  args.sigma = sigma;
  args.T = T;
  args.M = M;
  args.trials = trials;
  args.mean = 0.0;
  args.variance = 0.0;

    printf("00");

  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;

    printf("0");
  int *hostResults;
  int probsize=M;
  int nthreads=n;
  int nblocks=(probsize/nthreads)+1;

  printf("1");
  
  hostResults = (int *)calloc(nblocks * nthreads, sizeof(int));
    printf("2");
  hipMalloc((void **)&devResults, nblocks * nthreads * sizeof(int));
  hipMemset(devResults, 0, nblocks * nthreads *  sizeof(int));
  hipMalloc((void **)&devMTGPStates, nthreads * sizeof(hiprandStateMtgp32));
  hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));
    printf("3");
  hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
  hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, nthreads, 1234);

     printf("4"); 
  //args.devResults=devResults;
  //args.state=devMTGPStates;


  printf("5");
  (void)black_scholes_iterate<<<nblocks,nthreads>>>(&args);
    printf("6");
  mean = args.mean;
    printf("7");
  hipDeviceSynchronize();
  black_scholes_stddev<<<nblocks,nthreads>>> (&args);
    printf("8");
  hipDeviceSynchronize();
  conf_width = 1.96 * stddev / sqrt ((double) M);
    printf("9");
  interval->min = mean - conf_width;
    printf("10");
  interval->max = mean + conf_width;
    printf("11");
   hipMemcpy(hostResults, devResults, nblocks*blocks * sizeof(int), hipMemcpyDeviceToHost);
   hipFree(devMTGPStates);
   free(hostResults);
   hipFree(devResults);
   deinit_black_scholes_args (&args);
}


void deinit_black_scholes_args (black_scholes_args_t* args)
{
  if (args != NULL)
    if (args->trials != NULL)
      {
	free (args->trials);
	args->trials = NULL;
      }
}

